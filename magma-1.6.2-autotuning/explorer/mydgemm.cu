

#include <hip/hip_runtime.h>
#include<stdio.h>
#define NB 512

// encoding checksum for A

__global__ void
chkenc_kernel(double * A, int lda, double * Chk , int ldchk)
{
	//if (blockIdx.x == 0 && threadIdx.x == 0)
	//	printf("grid:%d, block:%d\n", gridDim.x, blockDim.x);
    //blockIdx.x: determin the column to process
	A = A + blockIdx.x * lda;

	__shared__ double cache[NB];
	
	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x];

	__syncthreads();

	int i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk) = cache[0];
	}


	//load one column to cache
	cache[threadIdx.x] = A[threadIdx.x] * (threadIdx.x + 1);

	__syncthreads();

	i = blockDim.x / 2;

	while (i != 0) {
		if (threadIdx.x < i)
			cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
		i /= 2;
	}

	if (threadIdx.x == 0) {
		*(Chk + blockIdx.x * ldchk + 1) = cache[0];
	}

	
}


void chkenc(double * A, int lda, int m, int n, double * Chk , int ldchk, hipStream_t stream) {
    int numBlocks; // Occupancy in terms of active blocks 
    int blockSize = 512; 
	int device; 
	hipDeviceProp_t prop; 
	int activeWarps; 
	int maxWarps; 
	hipGetDevice(&device); 
	hipGetDeviceProperties(&prop, device); hipOccupancyMaxActiveBlocksPerMultiprocessor( &numBlocks, chkenc_kernel, blockSize, 0); 
	activeWarps = numBlocks * blockSize / prop.warpSize; 
	maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize; 
	printf("Occupancy: %f \n", (double)activeWarps / maxWarps * 100 );

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(chkenc_kernel), hipFuncCachePreferShared);
	chkenc_kernel<<<n, m, 0, stream>>>(A, lda, Chk, ldchk);
}

int main(){
	int n = 30720;
	double * A = new double[NB * n];
	for (int i = 0; i < NB*n; i++) {
		A[i] = i;
	}
	double * dA;
	size_t dApitch;
	hipMallocPitch(&dA, &dApitch, NB*sizeof(double), n);
	hipMemcpy2D(dA, dApitch, A, NB, NB, n, hipMemcpyHostToDevice);
	int ldda = dApitch/sizeof(double);

	double * chk;
	size_t chkpitch;
	hipMallocPitch(&chk, &chkpitch, 2*sizeof(double), n);
	int ldchk = chkpitch/sizeof(double);

	hipStream_t stream;
	hipStreamCreate(&stream);

	chkenc(dA, ldda, NB, n, chk , ldchk, stream);





}



